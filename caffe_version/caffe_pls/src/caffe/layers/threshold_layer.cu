#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/threshold_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ThresholdForward(const int n, const Dtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > threshold ? in[index] : 0;
  }
}

template <typename Dtype>
void ThresholdLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ThresholdForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, threshold_, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void ThresholdBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype threshold_) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] > threshold_);
  }
}

template <typename Dtype>
void ThresholdLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();

    // NOLINT_NEXT_LINE(whitespace/operators)
    ThresholdBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, threshold_);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ThresholdLayer);


}  // namespace caffe
